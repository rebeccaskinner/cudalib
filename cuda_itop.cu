#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/types.h>

#ifdef __cplusplus
extern "C" {
#endif
__global__ void itop_kernel(const uint8_t* const in, uint8_t* out, const int data_len, const int thread_len)
{
    int offset = (blockDim.x * blockIdx.x * thread_len) + (blockIdx.x * thread_len);
    uint8_t* out_y = out + offset;
    uint8_t* out_b = out_y + (data_len / 3);
    uint8_t* out_r = out_b + (data_len / 3);
    for(int i = offset; i < thread_len + offset && i < data_len;)
    {
        *out_y++ = in[i++];
        *out_b++ = in[i++];
        *out_r++ = in[i++];
    }
}

__global__ void ptoi_kernel(const uint8_t * const in_y,
                            const uint8_t*  const in_b,
                            const uint8_t*  const in_r,
                            uint8_t* out, int data_len, int thread_len)
{
    int oset_main=(blockDim.x*blockIdx.x*thread_len)+(threadIdx.x*thread_len);
    for(int i=oset_main,j=oset_main;i<thread_len+oset_main&&i<data_len;++j)
    {
        out[i] = in_y[j];
        ++i;
        out[i] = in_b[j];
        ++i;
        out[i] = in_r[j];
        ++i;
    }
}

uint8_t* cuda_interstitial_to_planar(uint8_t* data, int data_len)
{
    if(data_len%3)
    {
        printf("Error: data_len must be a multiple of 3\n");
        return NULL;
    }
    int channel_len = data_len/3;
    int num_threads = channel_len/256;
    uint8_t* h_out = (uint8_t*)malloc(data_len);
    uint8_t* d_in, *d_out;
    hipMalloc((void**)&d_in,data_len);
    hipMalloc((void**)&d_out,data_len);
    hipMemcpy(d_in,data,data_len,hipMemcpyHostToDevice);
    printf("3 blocks and %d threads\n",num_threads);
    printf("thread len: %d\n",data_len/(3*num_threads));
    itop_kernel<<<3,num_threads>>>(d_in,d_out,data_len,data_len/(3*num_threads));
    hipMemcpy(h_out,d_out,data_len,hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
    return h_out;
}
#ifdef __cplusplus
}
#endif
